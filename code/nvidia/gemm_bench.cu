#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <memory>
#include <stdexcept>
#include <tuple>
#include <vector>
#include <cstdint>
#include <sstream>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hipblas.h>
#include <hiprand.h>

#include <thrust/device_ptr.h>
#include <thrust/fill.h>

#include "tensor.h"
#include "gemm_problems.h"

#ifndef PAD_KERNELS
#define PAD_KERNELS 1
#endif

#ifndef USE_TENSOR_CORES
#if __CUDACC_VER_MAJOR__ > 8
#define USE_TENSOR_CORES 1
#else
#define USE_TENSOR_CORES 0
#endif
#endif

/*
Usage:

The default precision is set based on the architecture and mode.

By default, the program runs the benchmark in training mode.

bin/gemm_bench

To run inference mode, use the following command:

bin/gemm_bench inference


To change the precision for training/inference, use:

bin/gemm_bench train <precision>
bin/gemm_bench inference <precision>

Supported precision types:

For Maxwell GPUS: 
float for training and inference

For Pascal/Volta GPUS:
float, half for training
float, half, int8 for inference

*/

template <typename T1, typename T2>
int time_gemm(Tensor<T1> A, Tensor<T1> B, Tensor<T2> C, bool a_t, bool b_t, hipblasHandle_t cublas_handle) {

#if (__CUDACC_VER_MAJOR__ >= 8)
    const int alpha = 1.f;
    const int beta  = 1.f;
#else
    const float alpha = 1.f / static_cast<float>(A.dims()[1]);
    const float beta  = 1.f;
#endif

    int m = C.dims()[0];
    int k = a_t ? A.dims()[0] : A.dims()[1];
    int n = C.dims()[1];

    int numRepeats = 400;
    hipblasStatus_t stat;

#if (__CUDACC_VER_MAJOR__ >= 8)
    hipDataType A_type = HIP_R_32F;
    hipDataType B_type = HIP_R_32F;
    hipDataType C_type = HIP_R_32F;
    hipDataType compute_type = HIP_R_32F;
    hipblasGemmAlgo_t algo;

    if (std::is_same<T1, __half>::value) {
        A_type = HIP_R_16F;
        B_type = HIP_R_16F;
    } 

    if (std::is_same<T2, float>::value) {
      C_type = HIP_R_32F;
      compute_type = HIP_R_32F;
    } else if (std::is_same<T2, __half>::value) {
      C_type = HIP_R_16F;
      compute_type = HIP_R_16F;
    } else if (std::is_same<T2, int>::value) {
      compute_type = HIP_R_32I;
    } else {
      std::cerr << "Unsuported T2 (output) type" << std::endl;
      exit(1);
    }

    if (std::is_same<T1, uint8_t>::value) {
        A_type = HIP_R_8I;
        B_type = HIP_R_8I;
        C_type = HIP_R_32I;
        compute_type = HIP_R_32I;
    }

#if (USE_TENSOR_CORES)
        algo = CUBLAS_GEMM_DFALT_TENSOR_OP;
#else
        algo = HIPBLAS_GEMM_DEFAULT;
#endif

#endif

#if (__CUDACC_VER_MAJOR__ < 8)
    // Warm up
    stat = hipblasSgemm(cublas_handle,
                a_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                b_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                m,
                n,
                k,
                &alpha,
                A.begin(), A.dims()[0],
                B.begin(), B.dims()[0],
                &beta,
                C.begin(), C.dims()[0]);
#else
    stat = hipblasGemmEx(cublas_handle,
                a_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                b_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                m,
                n,
                k,
                &alpha,
                A.begin(), A_type, A.dims()[0],
                B.begin(), B_type, B.dims()[0],
                &beta,
                C.begin(), C_type, C.dims()[0],
                compute_type,
                algo);
#endif

    if (stat != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("sgemm failed");
    }

    hipDeviceSynchronize();

    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < numRepeats; ++i) {

#if (__CUDACC_VER_MAJOR__ < 8)
        stat = hipblasSgemm(cublas_handle,
                    a_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                    b_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                    m,
                    n,
                    k,
                    &alpha,
                    A.begin(), A.dims()[0],
                    B.begin(), B.dims()[0],
                    &beta,
                    C.begin(), C.dims()[0]);
#else
        stat = hipblasGemmEx(cublas_handle,
                    a_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                    b_t ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                    m,
                    n,
                    k,
                    &alpha,
                    A.begin(), A_type, A.dims()[0],
                    B.begin(), B_type, B.dims()[0],
                    &beta,
                    C.begin(), C_type, C.dims()[0],
                    compute_type,
                    algo);
#endif
        if (stat != HIPBLAS_STATUS_SUCCESS) {
            throw std::runtime_error("sgemm failed");
        }
    }
    hipDeviceSynchronize();

    auto end = std::chrono::steady_clock::now();

    return static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() / numRepeats);

}

int main(int argc, char **argv) {
    hipFree(0);

    int inference = 0;
    if (argc > 1) {
        std::string inf = "inference";
        inference = argv[1] == inf ? 1 : 0;
    }

#if (__CUDACC_VER_MAJOR__ >= 8)
    std::string precision;
    if (inference)
        precision = "int8";
    else
        precision = "half";
#else
    std::string precision = "float";
#endif
    if (argc > 2) {
        precision = argv[2];
    }

    hipblasHandle_t cublas_handle;
    hipblasStatus_t status = hipblasCreate(&cublas_handle);
    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "CUBLAS init failed" << std::endl;
    }

#if (USE_TENSOR_CORES) && (__CUDACC_VER_MAJOR__ > 8)
    status = hipblasSetMathMode(cublas_handle, HIPBLAS_TENSOR_OP_MATH);
#endif

    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "CUBLAS math mode failed" << std::endl;
    } else std::cout << "CUBALS_TENSOR_OP_MATH ON" << std::endl;


    hiprandGenerator_t curand_gen;

    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 123ULL);

    if (inference) {
        std::cout << std::setw(45) << "Running inference benchmark " << std::endl;
    } else {
        std::cout << std::setw(45) << "Running training benchmark " << std::endl;
    }

    std::cout << std::setw(30) << "Times" << std::endl;
    std::cout << std::setfill('-') << std::setw(88) << "-" << std::endl;
    std::cout << std::setfill(' ');
    std::cout << "    m       n      k      a_t     b_t      precision        time (usec) ";

    if (PAD_KERNELS && precision == "int8" && inference)
        std::cout << " pad_kerenels  ";


    std::cout << std::endl;

    int pad_kernels_count = 0;

    for (const auto &problem : (inference ? inference_server_set : training_set)) {
        int m, n, k;
        bool a_t, b_t;
        std::tie(m, n, k, a_t, b_t) = problem;
        int time_ms;
        bool skip_kernel = false;
        bool need_padding = false;


#if (__CUDACC_VER_MAJOR__ >= 8)
        int pad_m;
        pad_m = m;
        if (precision == "int8") {
            if (pad_m%4) {
                pad_kernels_count++;
                if (PAD_KERNELS) {
                    pad_dim(pad_m, 4);
                    need_padding = true;
                } else {
                    skip_kernel = true;
                }
            }
        }
#endif

        std::cout << std::setw(7) << m;
        std::cout << std::setw(7) << n;
        std::cout << std::setw(7) << k;
        std::cout << std::setw(7) << a_t ? "t" : "n";
        std::cout << std::setw(7) << b_t ? "t" : "n";

        std::stringstream ss;
        ss << "Unsupported precision requested. Precision: " << precision << " Inference: " << inference;

#if (__CUDACC_VER_MAJOR__ >= 8)
        if (precision == "int8" & inference) {
            auto a = rand<uint8_t>({a_t ? k : pad_m, a_t ? pad_m : k}, curand_gen);
            auto b = rand<uint8_t>({b_t ? n : k, b_t ? k : n}, curand_gen);
            auto c = zeros<int>({pad_m, n});
            std::cout << std::setw(14) << precision;
            if (!skip_kernel)
                time_ms = time_gemm<uint8_t, int>(a, b, c, a_t, b_t, cublas_handle);
        } else if (precision == "half") {
            auto a = rand<__half>({a_t ? k : m, a_t ? m : k}, curand_gen);
            auto b = rand<__half>({b_t ? n : k, b_t ? k : n}, curand_gen);
            auto c = zeros<__half>({m, n});
            std::cout << std::setw(13) << precision;
            time_ms = time_gemm<__half, __half>(a, b, c, a_t, b_t, cublas_handle);
        } else if (precision == "float") {
            auto a = rand<float>({a_t ? k : m, a_t ? m : k}, curand_gen);
            auto b = rand<float>({b_t ? n : k, b_t ? k : n}, curand_gen);
            auto c = zeros<float>({m, n});
            std::cout << std::setw(13) << precision;
            time_ms = time_gemm<float, float>(a, b, c, a_t, b_t, cublas_handle);
        } else if (precision == "mixed") { // f16 x f16 to f32
            auto a = rand<__half>({a_t ? k : m, a_t ? m : k}, curand_gen);
            auto b = rand<__half>({b_t ? n : k, b_t ? k : n}, curand_gen);
            auto c = zeros<float>({m, n});
            std::cout << std::setw(13) << precision;
            time_ms = time_gemm<__half, float>(a, b, c, a_t, b_t, cublas_handle);
	} else {
            throw std::runtime_error(ss.str());
        }
#else

        if (precision != "float") {
            throw std::runtime_error(ss.str());
        }

        auto a = rand<float>({a_t ? k : m, a_t ? m : k}, curand_gen);
        auto b = rand<float>({b_t ? n : k, b_t ? k : n}, curand_gen);
        auto c = zeros<float>({m, n});
        std::cout << std::setw(13) << precision;
        time_ms = time_gemm<float, float>(a, b, c, a_t, b_t, cublas_handle);
#endif
        std::cout << std::setw(20) << std::setprecision(6);

        if (skip_kernel) {
            std::cout << "Not Supported";
        } else {
            std::cout << time_ms;
        }

        if (PAD_KERNELS && precision == "int8" && inference) {
            std::cout << std::setw(10) <<  need_padding;
        }

        std::cout << std::endl;
    }

    if (precision == "int8") {
        std::cout << " Total kernels ";
        if (PAD_KERNELS)
            std::cout << "padded: " << pad_kernels_count << std::endl;
        else
            std::cout << "skipped: " << pad_kernels_count << std::endl;

        std::cout << " Total kernels: " << inference_server_set.size() << std::endl;
    }

    hipblasDestroy(cublas_handle);
    hiprandDestroyGenerator(curand_gen);

    return 0;
}
